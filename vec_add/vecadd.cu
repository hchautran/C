#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>

using namespace std;


__global__ void vecadd_kernel(float* a, float *b,  float* c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
       c[idx] = a[idx] + b[idx]; 
    } 
}


void printArr(float* c, int n) {
    for (int i=0; i<n; i++) {
        printf("%.2f ", c[i]);
    }
    printf("\n");
}

void vecadd(float* a_h, float* b_h, float* c_h, int n) {
    int size = n * sizeof(float);
    float *a_d, *b_d, *c_d;
    hipError_t cudaStatus = hipMalloc((void**)&a_d, size);
    if (cudaStatus != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    cudaStatus = hipMalloc((void**)&b_d, size);
    if (cudaStatus != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    cudaStatus = hipMalloc((void**)&c_d, size);
    if (cudaStatus != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cudaStatus),__FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);


    int n_threads = 256;
    int n_blocks = ceil(n/256.0);
    vecadd_kernel<<<n_threads, n_blocks>>>(a_d, b_d, c_d, n);
    hipDeviceSynchronize();
    
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

}


int main() {
    int n = 5;
    float a[n] = {1., 2., 3., 4., 5.};
    float b[n] = {1., 2., 3., 4., 5.};
    float c[n] = {0.,0.,0.,0.,0.};
    printArr(c, n);
    vecadd(a, b, c, n);
    printArr(c, n);
    return 0;
}